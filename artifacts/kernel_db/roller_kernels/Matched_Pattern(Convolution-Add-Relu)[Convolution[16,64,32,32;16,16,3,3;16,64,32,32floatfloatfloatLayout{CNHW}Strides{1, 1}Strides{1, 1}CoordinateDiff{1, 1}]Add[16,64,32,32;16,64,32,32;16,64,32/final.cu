
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ bias, float* __restrict__ conv_unpad) {
  float conv_local[2];
  __shared__ float data_pad_shared[8192];
  __shared__ float kernel_pad_shared[2048];
  float data_pad_shared_local[2];
  float kernel_pad_shared_local[1];
  conv_local[(0)] = 0.000000e+00f;
  conv_local[(1)] = 0.000000e+00f;
  for (int ra_fused0_outer = 0; ra_fused0_outer < 2; ++ra_fused0_outer) {
    __syncthreads();
    data_pad_shared[(((int)threadIdx.x))] = (((((0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 9) / 3))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)))) && (((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) < 33)) ? data[((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) / 9) * 65536) + (((int)blockIdx.x) * 64)) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 512))] = (((((0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) % 9) / 3))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 1024))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 128) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 7) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 7) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 16) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 7) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 1536))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 120) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 6) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 6) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)))) && (((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 24) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 6) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 2048))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 112) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 5) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 5) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 32) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 5) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 2560))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 104) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 4) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 4) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 40) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 4) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 3072))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 96) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 3) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 3) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)))) && (((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 48) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 3) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 3584))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 88) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 56) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 4096))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 80) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 64) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 4608))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 72) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)))) && (((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) < 33)) ? data[((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) / 9) * 65536) + (((int)blockIdx.x) * 64)) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) + 524255))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 5120))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 64) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 80) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 8) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 5632))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 56) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 7) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 7) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 88) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 7) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 6144))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 48) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 6) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 6) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)))) && (((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 96) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 6) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 6656))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 40) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 5) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 5) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 104) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 5) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 2) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 7168))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 32) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 4) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 4) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)))) && (((((int)threadIdx.x) & 31) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 112) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 4) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + ((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 1) % 3)) - 33))] : 0.000000e+00f);
    data_pad_shared[((((int)threadIdx.x) + 7680))] = ((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) < 24) && (0 < (((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 3) % 9) / 3)))) && ((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)) & 1023) >> 5) + (((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 3) % 9) / 3)) < 33)) && (0 < ((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)))) && (((((int)threadIdx.x) & 31) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) < 33)) ? data[(((((((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 120) / 9) * 65536) + (((int)blockIdx.x) * 64)) + ((((((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) + 3) % 9) / 3) * 32)) + (((int)threadIdx.x) & 63)) + (((ra_fused0_outer * 128) + (((int)threadIdx.x) >> 6)) % 3)) - 33))] : 0.000000e+00f);
    kernel_pad_shared[(((int)threadIdx.x))] = ((((ra_fused0_outer * 128) + (((int)threadIdx.x) & 127)) < 144) ? kernel[(((((((int)threadIdx.x) >> 7) * 144) + (ra_fused0_outer * 128)) + (((int)threadIdx.x) & 127)))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 512))] = ((((ra_fused0_outer * 128) + (((int)threadIdx.x) & 127)) < 144) ? kernel[((((((((int)threadIdx.x) >> 7) * 144) + (ra_fused0_outer * 128)) + (((int)threadIdx.x) & 127)) + 576))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 1024))] = ((((ra_fused0_outer * 128) + (((int)threadIdx.x) & 127)) < 144) ? kernel[((((((((int)threadIdx.x) >> 7) * 144) + (ra_fused0_outer * 128)) + (((int)threadIdx.x) & 127)) + 1152))] : 0.000000e+00f);
    kernel_pad_shared[((((int)threadIdx.x) + 1536))] = ((((ra_fused0_outer * 128) + (((int)threadIdx.x) & 127)) < 144) ? kernel[((((((((int)threadIdx.x) >> 7) * 144) + (ra_fused0_outer * 128)) + (((int)threadIdx.x) & 127)) + 1728))] : 0.000000e+00f);
    __syncthreads();
    for (int ra_fused0_inner_outer = 0; ra_fused0_inner_outer < 128; ++ra_fused0_inner_outer) {
      data_pad_shared_local[(0)] = data_pad_shared[(((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 31)))];
      data_pad_shared_local[(1)] = data_pad_shared[((((ra_fused0_inner_outer * 64) + (((int)threadIdx.x) & 31)) + 32))];
      kernel_pad_shared_local[(0)] = kernel_pad_shared[((((((int)threadIdx.x) >> 5) * 128) + ra_fused0_inner_outer))];
      if (((ra_fused0_outer * 128) + ra_fused0_inner_outer) < 144) {
        conv_local[(0)] = (conv_local[(0)] + (data_pad_shared_local[(0)] * kernel_pad_shared_local[(0)]));
        conv_local[(1)] = (conv_local[(1)] + (data_pad_shared_local[(1)] * kernel_pad_shared_local[(0)]));
      }
    }
  }
  conv_unpad[(((((((int)threadIdx.x) >> 5) * 65536) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 31)))] = max((conv_local[(0)] + bias[((((int)threadIdx.x) >> 5))]), 0.000000e+00f);
  conv_unpad[((((((((int)threadIdx.x) >> 5) * 65536) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 31)) + 32))] = max((conv_local[(1)] + bias[((((int)threadIdx.x) >> 5))]), 0.000000e+00f);
}

dim3 grid(1024, 1, 1);
dim3 block(512, 1, 1);